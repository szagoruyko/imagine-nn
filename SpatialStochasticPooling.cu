#include "hip/hip_runtime.h"
#include <THC/THC.h>
#include "common.h"

// kernels borrowed from Caffe

__global__ void StoPoolForwardTrain(const int nthreads,
    const float* bottom_data,
    const int num, const int channels, const int height,
    const int width, const int pooled_height, const int pooled_width,
    const int kernel_h, const int kernel_w, const int stride_h,
    const int stride_w, float* rand_idx, float* top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;
    int hstart = ph * stride_h;
    int hend = min(hstart + kernel_h, height);
    int wstart = pw * stride_w;
    int wend = min(wstart + kernel_w, width);
    float cumsum = 0.;
    bottom_data += (n * channels + c) * height * width;
    // First pass: get sum
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        cumsum += bottom_data[h * width + w];
      }
    }
    float thres = rand_idx[index] * cumsum;
    // Second pass: get value, and set index.
    cumsum = 0;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        cumsum += bottom_data[h * width + w];
        if (cumsum >= thres) {
          rand_idx[index] = ((n * channels + c) * height + h) * width + w;
          top_data[index] = bottom_data[h * width + w];
          return;
        }
      }
    }
  }
}


__global__ void StoPoolForwardTest(const int nthreads,
    const float* bottom_data,
    const int num, const int channels, const int height,
    const int width, const int pooled_height, const int pooled_width,
    const int kernel_h, const int kernel_w, const int stride_h,
    const int stride_w, float* top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;
    int hstart = ph * stride_h;
    int hend = min(hstart + kernel_h, height);
    int wstart = pw * stride_w;
    int wend = min(wstart + kernel_w, width);
    // We set cumsum to be 0 to avoid divide-by-zero problems
    float cumsum = FLT_MIN;
    float cumvalues = 0.;
    bottom_data += (n * channels + c) * height * width;
    // First pass: get sum
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        cumsum += bottom_data[h * width + w];
        cumvalues += bottom_data[h * width + w] * bottom_data[h * width + w];
      }
    }
    top_data[index] = cumvalues / cumsum;
  }
}

extern "C"
void SpatialStochasticPooling_updateOutput(THCState* state, THCudaTensor* input, 
    THCudaTensor* output, THCudaTensor* indices, int kW, int kH, int dW, int dH, bool train)
{
  long nInputCols, nInputRows, nInputPlane, batchSize;

  if (input->nDimension == 3) {
    nInputCols = input->size[2];
    nInputRows = input->size[1];
    nInputPlane = input->size[0];
    batchSize = 1;
  }
  else
  {
    nInputCols = input->size[3];
    nInputRows = input->size[2];
    nInputPlane = input->size[1];
    batchSize = input->size[0];
  }

  long nOutputCols = ceil(float(nInputCols - kW) / float(dW)) + 1;
  long nOutputRows = ceil(float(nInputRows - kH) / float(dH)) + 1;

  input = THCudaTensor_newContiguous(state, input);
  float* input_data = THCudaTensor_data(state, input);

  THCudaTensor_resize4d(state, output, batchSize, nInputPlane, nOutputRows, nOutputCols);
  THCudaTensor_resizeAs(state, indices, output);
  THCudaTensor_uniform(state, indices, 0, 1);
  
  float* indices_data = THCudaTensor_data(state, indices);
  float* output_data = THCudaTensor_data(state, output);

  int count = THCudaTensor_nElement(state, output);

  if(train)
    StoPoolForwardTrain <<< GET_BLOCKS(count), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state) >>>
      	(count, input_data,
	batchSize, nInputPlane, nInputRows, nInputCols, nOutputRows, nOutputCols,
	kH, kW, dH, dW, indices_data, output_data);
  else
    StoPoolForwardTest <<< GET_BLOCKS(count), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state) >>>
      	(count, input_data,
	batchSize, nInputPlane, nInputRows, nInputCols, nOutputRows, nOutputCols,
	kH, kW, dH, dW, output_data);

  if(input->nDimension == 3)
    THCudaTensor_resize3d(state, output, nInputPlane, nOutputRows, nOutputCols);
  THCudaTensor_free(state, input);
}


__global__ void StoPoolBackward(const int nthreads,
    const float* rand_idx, const float* top_diff,
    const int num, const int channels, const int height,
    const int width, const int pooled_height, const int pooled_width,
    const int kernel_h, const int kernel_w, const int stride_h,
    const int stride_w, float* bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    int w = index % width;
    int h = (index / width) % height;
    int c = (index / width / height) % channels;
    int n = index / width / height / channels;
    int phstart = (h < kernel_h) ? 0 : (h - kernel_h) / stride_h + 1;
    int phend = min(h / stride_h + 1, pooled_height);
    int pwstart = (w < kernel_w) ? 0 : (w - kernel_w) / stride_w + 1;
    int pwend = min(w / stride_w + 1, pooled_width);
    float gradient = 0;
    rand_idx += (n * channels + c) * pooled_height * pooled_width;
    top_diff += (n * channels + c) * pooled_height * pooled_width;
    for (int ph = phstart; ph < phend; ++ph) {
      for (int pw = pwstart; pw < pwend; ++pw) {
        gradient += top_diff[ph * pooled_width + pw] *
            (index == static_cast<int>(rand_idx[ph * pooled_width + pw]));
      }
    }
    bottom_diff[index] = gradient;
  }
}

extern "C"
void SpatialStochasticPooling_updateGradInput(THCState* state, THCudaTensor* input,
    THCudaTensor* gradInput, THCudaTensor* gradOutput, THCudaTensor* indices, int kW, int kH, int dW, int dH)
{
  long nInputCols, nInputRows, nInputPlane, batchSize;

  if (input->nDimension == 3) {
    nInputCols = input->size[2];
    nInputRows = input->size[1];
    nInputPlane = input->size[0];
    batchSize = 1;
  }
  else
  {
    nInputCols = input->size[3];
    nInputRows = input->size[2];
    nInputPlane = input->size[1];
    batchSize = input->size[0];
  }

  long nOutputCols = ceil(float(nInputCols - kW) / float(dW)) + 1;
  long nOutputRows = ceil(float(nInputRows - kH) / float(dH)) + 1;

  gradOutput = THCudaTensor_newContiguous(state, gradOutput);
  THCudaTensor_resizeAs(state, gradInput, input);
  
  int count = THCudaTensor_nElement(state, input);

  StoPoolBackward <<< GET_BLOCKS(count), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state) >>> 
      (count,
      THCudaTensor_data(state, indices),
      THCudaTensor_data(state, gradOutput),
      batchSize, nInputPlane, nInputRows, nInputCols, nOutputRows, nOutputCols,
      kH, kW, dH, dW,
      THCudaTensor_data(state, gradInput));

  THCudaTensor_free(state, gradOutput);
}
