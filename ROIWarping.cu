#include "hip/hip_runtime.h"
// ------------------------------------------------------------------
// Fast R-CNN
// Copyright (c) 2015 Microsoft
// Licensed under The MIT License [see fast-rcnn/LICENSE for details]
// Written by Ross Girshick
// ------------------------------------------------------------------

// Torch port:
// IMAGINE, Sergey Zagoruyko, Francisco Massa, 2015

#include "THC.h"
#include <algorithm>
#include <cfloat>
#include "assert.h"

#include "common.h"


using std::max;
using std::min;


template <typename Dtype>
__global__ void ROIWarpForward(const int nthreads, const Dtype* bottom_data,
    const Dtype spatial_scale, const int channels, const int height,
    const int width, const int pooled_height, const int pooled_width,
    const Dtype* bottom_rois, const Dtype* bottom_delta_rois, Dtype* top_data, Dtype* top_data_buffer) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // (n, c, ph, pw) is an element in the pooled output
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;

    bottom_rois += n * 5;
    int roi_batch_ind = (bottom_rois[0] - 1);
    //int roi_start_w = round((bottom_rois[1] - 1) * spatial_scale);
    //int roi_start_h = round((bottom_rois[2] - 1)* spatial_scale);
    //int roi_end_w = round((bottom_rois[3] - 1) * spatial_scale);
    //int roi_end_h = round((bottom_rois[4] - 1) * spatial_scale);

    Dtype src_w = bottom_rois[3] - bottom_rois[1] + 1; 
    Dtype src_h = bottom_rois[4] - bottom_rois[2] + 1;
    Dtype src_ctr_x = bottom_rois[1] + 0.5*(src_w-1.0); 
    Dtype src_ctr_y = bottom_rois[2] + 0.5*(src_h-1.0); 

    Dtype dst_ctr_x = bottom_delta_rois[1]; // dx (in fast-rcnn notation) = cx (in here)
    Dtype dst_ctr_y = bottom_delta_rois[2]; // dy (in fast-rcnn notation) = cy (in here) 
    Dtype dst_scl_x = bottom_delta_rois[3]; // dw (in fast-rcnn notation) = sx (in here)
    Dtype dst_scl_y = bottom_delta_rois[4]; // dh (in fast-rcnn notation) = sy (in here) 

    Dtype pred_ctr_x = dst_ctr_x * src_w + src_ctr_x; 
    Dtype pred_ctr_y = dst_ctr_y * src_h + src_ctr_y; 
    Dtype pred_w = exp(dst_scl_x) * src_w;            
    Dtype pred_h = exp(dst_scl_y) * src_h;            
    
    Dtype roi_start_w = ( (pred_ctr_x - 0.5*(pred_w-1)) - 1 ) * spatial_scale; 
    Dtype roi_start_h = ( (pred_ctr_y - 0.5*(pred_h-1)) - 1 ) * spatial_scale; 
    Dtype roi_end_w =   ( (pred_ctr_x + 0.5*(pred_w-1)) - 1 ) * spatial_scale; 
    Dtype roi_end_h =   ( (pred_ctr_y + 0.5*(pred_h-1)) - 1 ) * spatial_scale; 
    assert(roi_end_w - roi_start_w >= 0);
    assert(roi_end_h - roi_start_h >= 0);   
    
    // Force malformed ROIs to be 1x1
    Dtype roi_width  = roi_end_w - roi_start_w + 1; 
    Dtype roi_height = roi_end_h - roi_start_h + 1; 
    
    Dtype bin_size_w = roi_width  / static_cast<Dtype>(pooled_width);  
    Dtype bin_size_h = roi_height / static_cast<Dtype>(pooled_height); 
    Dtype wstart_ = static_cast<Dtype>(pw) * bin_size_w + roi_start_w;
    Dtype hstart_ = static_cast<Dtype>(ph) * bin_size_h + roi_start_h;
    Dtype wend_ = static_cast<Dtype>(pw+1) * bin_size_w + roi_start_w;
    Dtype hend_ = static_cast<Dtype>(ph+1) * bin_size_h + roi_start_h;
                                                                       
    int wstart = static_cast<int>(floor(wstart_)); 
    int hstart = static_cast<int>(floor(hstart_)); 
    int wend   = static_cast<int>( ceil(wend_)); 
    int hend   = static_cast<int>( ceil(hend_));
 
    Dtype wctr =  (wend_ + wstart_) * 0.5; // dwctr / dwe = 0.5; dwctr / dws = 0.5
    Dtype hctr =  (hend_ + hstart_) * 0.5; // dhctr / dhe = 0.5; dhctr / dhs = 0.5
    Dtype wdiff = (wend_ - wstart_) + 1;   // dwdiff / dwe = 1; dwdiff / dws = -1
    Dtype hdiff = (hend_ - hstart_) + 1;   // dhdiff / dhe = 1; dhdiff / dhs = -1

    //top_data[index] = static_cast<Dtype>(hend-1-hstart)+1;
    //top_data[index] = hend; //wend;
    //top_data[index] = hstart+1; // wstart+1;
    //top_data[index] = wdiff;
    //top_data[index] = hctr+1;
    //top_data[index] = wctr+1;
   
    // Add roi offsets and clip to input boundaries
    hstart = min(max(hstart, 0), height);
    hend   = min(max(hend, 0), height);
    wstart = min(max(wstart, 0), width);
    wend   = min(max(wend, 0), width);

    //top_data[index] = hstart+1; 
    //top_data[index] = wstart+1;

    // Auxilliary variables used in backprop 
    Dtype w_mask = 0, h_mask = 0; 
    Dtype dgx_final_dwctr_all  = 0;
    Dtype dgx_final_dwdiff_all = 0;
    Dtype dgy_final_dhctr_all  = 0;
    Dtype dgy_final_dhdiff_all = 0; 
    // Define an empty pooling region to be zero
    Dtype val = 0; Dtype gain = 0, gain_x = 0, gain_y = 0, gain_x_all = 0, gain_y_all = 0;   
    bottom_data += (roi_batch_ind * channels + c) * height * width;
    for (int h = hstart; h < hend; ++h) {
      Dtype h_ = h; 
      for (int w = wstart; w < wend; ++w) {
        int bottom_index = h * width + w;
        Dtype w_ = w;  
        //gain_x = max(0., 1 - abs( dst_ctr_x + static_cast<Dtype>(pw) / static_cast<Dtype>(pooled_width) * dst_scl_x - w )); -- in paper, but makes no sense
        //gain_y = max(0., 1 - abs( dst_ctr_y + static_cast<Dtype>(ph) / static_cast<Dtype>(pooled_height) * dst_scl_y - h));
        gain_x = wdiff - abs((w_ - wctr)); 
        gain_y = hdiff - abs((h_ - hctr));   
        gain = gain_x * gain_y;

        val = val + gain * bottom_data[bottom_index];
        //val = val + gain;
        //val = val + 1;

        if (h == hstart) { 
          gain_x_all = gain_x_all + gain_x;

          // Update information used in backprop
          w_mask = w_ >= wctr ? 1 : -1;
          dgx_final_dwctr_all  = dgx_final_dwctr_all  + w_mask;
          dgx_final_dwdiff_all = dgx_final_dwdiff_all + 1;
        }
      }
      gain_y_all = gain_y_all + gain_y;
        
      h_mask = h >= hctr ? 1 : -1;
      dgy_final_dhctr_all  = dgy_final_dhctr_all  + h_mask;
      dgy_final_dhdiff_all = dgy_final_dhdiff_all + 1;
    }
    if (gain_x_all > 1e-10)
      val = val / gain_x_all;
    if (gain_y_all > 1e-10)  
      val = val / gain_y_all;
    top_data[index] = val;

    //top_data[index] = gain_x_all; 
    //top_data[index] = gain_y_all; 
    int buffer_index = n * (channels * pooled_height * pooled_width * 10) + c * (pooled_height * pooled_width * 10) + ph * (pooled_width * 10) + pw * 10;
    top_data_buffer[buffer_index+0] = wctr;
    top_data_buffer[buffer_index+1] = wdiff;
    top_data_buffer[buffer_index+2] = hctr;
    top_data_buffer[buffer_index+3] = hdiff; 
    top_data_buffer[buffer_index+4] = gain_x_all; 
    top_data_buffer[buffer_index+5] = gain_y_all;
    top_data_buffer[buffer_index+6] = dgx_final_dwctr_all;
    top_data_buffer[buffer_index+7] = dgy_final_dhctr_all;
    top_data_buffer[buffer_index+8] = dgx_final_dwdiff_all;
    top_data_buffer[buffer_index+9] = dgy_final_dhdiff_all;
  }
}

extern "C"
void inn_ROIWarping_updateOutput(THCState *state,
    THCudaTensor *output, THCudaTensor *output_buffer,
    THCudaTensor *data, THCudaTensor* rois, THCudaTensor* delta_rois, int W, int H, double spatial_scale)
{
  THAssert(THCudaTensor_nDimension(state, data) == 4);
  THAssert(THCudaTensor_nDimension(state, rois) == 2 && rois->size[1] == 5);
  THAssert(THCudaTensor_nDimension(state, delta_rois) == 2 && delta_rois->size[1] == 5);
  THAssert(THCudaTensor_nDimension(state, rois) == THCudaTensor_nDimension(state, delta_rois) &&
           rois->size[0] == delta_rois->size[0] &&
           rois->size[1] == delta_rois->size[1]);
  THAssert(THCudaTensor_isContiguous(state, data));
  THAssert(THCudaTensor_isContiguous(state, rois));
  THAssert(THCudaTensor_isContiguous(state, delta_rois));
  long num_rois = rois->size[0];
  long nInputPlane = data->size[1];
  THCudaTensor_resize4d(state, output, num_rois, nInputPlane, H, W);
  THCudaTensor_resize5d(state, output_buffer, num_rois, nInputPlane, H, W, 10);
  THCudaTensor_zero(state, output_buffer);

  long count = THCudaTensor_nElement(state, output);

  ROIWarpForward<float><<<GET_BLOCKS(count), CUDA_NUM_THREADS, 0, THCState_getCurrentStream(state)>>>(
      count,
      THCudaTensor_data(state, data),
      spatial_scale, nInputPlane, data->size[2], data->size[3], H, W,
      THCudaTensor_data(state, rois),
      THCudaTensor_data(state, delta_rois),
      THCudaTensor_data(state, output),
      THCudaTensor_data(state, output_buffer)
      );

  // check for errors
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in inn_ROIWarping_updateOutput: %s\n", hipGetErrorString(err));
    THError("aborting");
  }
}

template <typename Dtype>
__global__ void ROIWarpBackwardData(const int nthreads, const Dtype* top_data_buffer,
    const Dtype spatial_scale, const int channels, const int height, const int width, 
    const int pooled_height, const int pooled_width, const int nth_roi, 
    const Dtype* bottom_rois, const Dtype* bottom_delta_rois, 
    const Dtype* top_diff,
    Dtype* bottom_diff_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {

    // (n, c, h, w) is an element in the input 
    int w = index % width;
    int h = (index / width) % height;
    int c = (index / width / height) % channels;
    int n = index / width / height / channels;

    bottom_rois += nth_roi * 5;
    int roi_batch_ind = (bottom_rois[0] - 1);
 
    if (roi_batch_ind == n) {

      Dtype src_w = bottom_rois[3] - bottom_rois[1] + 1;
      Dtype src_h = bottom_rois[4] - bottom_rois[2] + 1;
      Dtype src_ctr_x = bottom_rois[1] + 0.5*(src_w-1.0);
      Dtype src_ctr_y = bottom_rois[2] + 0.5*(src_h-1.0);
  
      Dtype dst_ctr_x = bottom_delta_rois[1]; // dx (in fast-rcnn notation) = cx (in here)
      Dtype dst_ctr_y = bottom_delta_rois[2]; // dy (in fast-rcnn notation) = cy (in here)
      Dtype dst_scl_x = bottom_delta_rois[3]; // dw (in fast-rcnn notation) = sx (in here)
      Dtype dst_scl_y = bottom_delta_rois[4]; // dh (in fast-rcnn notation) = sy (in here)
  
      Dtype pred_ctr_x = dst_ctr_x * src_w + src_ctr_x; // dpcx / dcx = src_w
      Dtype pred_ctr_y = dst_ctr_y * src_h + src_ctr_y; // dpcy / dcy = src_h
      Dtype pred_w = exp(dst_scl_x) * src_w;            // dpw  / dsx = src_w * exp(dsx)
      Dtype pred_h = exp(dst_scl_y) * src_h;            // dph  / dsy = src_h * exp(dsy)
  
      Dtype roi_start_w = ( (pred_ctr_x - 0.5*(pred_w-1)) - 1 ) * spatial_scale; // drsw / dpcx = spatial_scale; drsw / dpw = -0.5 * spatial_scale
      Dtype roi_start_h = ( (pred_ctr_y - 0.5*(pred_h-1)) - 1 ) * spatial_scale; // drsh / dpcy = spatial_scale; drsh / dph = -0.5 * spatial_scale
      Dtype roi_end_w =   ( (pred_ctr_x + 0.5*(pred_w-1)) - 1 ) * spatial_scale; // drew / dpcx = spatial_scale; drew / dpw =  0.5 * spatial_scale
      Dtype roi_end_h =   ( (pred_ctr_y + 0.5*(pred_h-1)) - 1 ) * spatial_scale; // dreh / dpcy = spatial_scale; dreh / dph =  0.5 * spatial_scale
      assert(roi_end_w - roi_start_w >= 0);
      assert(roi_end_h - roi_start_h >= 0);
  
      Dtype roi_width  = roi_end_w - roi_start_w + 1;
      Dtype roi_height = roi_end_h - roi_start_h + 1;
  
      Dtype bin_size_pw = static_cast<Dtype>(pooled_width)  / roi_width;  
      Dtype bin_size_ph = static_cast<Dtype>(pooled_height) / roi_height; 
  
      int pwstart = static_cast<int>(floor(static_cast<Dtype>(-roi_start_w + w) * bin_size_pw)); 
      int phstart = static_cast<int>(floor(static_cast<Dtype>(-roi_start_h + h) * bin_size_ph)); 
      int pwend = static_cast<int>(ceil(static_cast<Dtype>(-roi_start_w + w+1) * bin_size_pw));
      int phend = static_cast<int>(ceil(static_cast<Dtype>(-roi_start_h + h+1) * bin_size_ph)); 
   
      //bottom_diff_data[index] = pwend; //phend; 
      //bottom_diff_data[index] = pwstart+1; //phend; 
  
      // Clip to top boundaries
      phstart = min(max(phstart, 0), pooled_height);         
      phend =   min(max(phend, 0),   pooled_height);
      pwstart = min(max(pwstart, 0), pooled_width);
      pwend =   min(max(pwend, 0),   pooled_width);
  
      Dtype w_ = w, h_ = h; 
      Dtype wctr = 0, wdiff = 0, hctr = 0, hdiff = 0;
      Dtype gain = 0, gain_x = 0, gain_y = 0, gain_x_all = 0, gain_y_all = 0;  
      for (int ph = phstart; ph < phend; ++ph) {
        for (int pw = pwstart; pw < pwend; ++pw) {
          int top_index = nth_roi * (channels * pooled_height * pooled_width) + c * (pooled_height * pooled_width) + ph * pooled_width  + pw;
          int top_buffer_index = nth_roi * (channels * pooled_height * pooled_width * 10) + c * (pooled_height * pooled_width * 10) + ph * (pooled_width * 10) + pw * 10;
          wctr       = top_data_buffer[top_buffer_index+0]; 
          wdiff      = top_data_buffer[top_buffer_index+1]; 
          hctr       = top_data_buffer[top_buffer_index+2]; 
          hdiff      = top_data_buffer[top_buffer_index+3]; 
          gain_x_all = top_data_buffer[top_buffer_index+4]; 
          gain_y_all = top_data_buffer[top_buffer_index+5]; 
  
          gain_x = wdiff - abs((w_ - wctr));   // dgx / dwdiff =   1  
                                               // dgx / dwctr  =   1 ( if w >= wctr )
                                               // dgx / dwctr  = - 1 ( else )
          gain_y = hdiff - abs((h_ - hctr));   // dgy / dhdiff =   1
                                               // dgy / dhctr  =   1 ( if h >= hctr )
                                               // dgy / dhctr  = - 1 ( else )
          if (gain_x_all > 1e-10) 
            gain_x = gain_x / gain_x_all; 
          if (gain_y_all > 1e-10)  
            gain_y = gain_y / gain_y_all; 
  
          gain = gain_x * gain_y;
          bottom_diff_data[index] = bottom_diff_data[index] + gain * top_diff[top_index]; //val = val + gain * bottom_data[bottom_index];
        }
      }
    }
  }
}

template <typename Dtype>
__global__ void ROIWarpBackwardDeltaROI(const int nthreads, const Dtype* top_data_buffer,
    const Dtype spatial_scale, const int channels, const int height, const int width,
    const int pooled_height, const int pooled_width, 
    const Dtype* bottom_rois, const Dtype* bottom_delta_rois,
    const Dtype* top_diff,
    const Dtype* bottom_data,
    Dtype* bottom_diff_delta_rois_buffer) {
  CUDA_KERNEL_LOOP(index, nthreads) { 
    // (n, c, ph, pw) is an element in the pooled output
    int pw = index % pooled_width;
    int ph = (index / pooled_width) % pooled_height;
    int c = (index / pooled_width / pooled_height) % channels;
    int n = index / pooled_width / pooled_height / channels;

    int buffer_index = n * (channels * pooled_height * pooled_width * 10) + c * (pooled_height * pooled_width * 10) + ph * (pooled_width * 10) + pw * 10; 
    Dtype wctr                 = top_data_buffer[buffer_index+0];
    Dtype wdiff                = top_data_buffer[buffer_index+1];
    Dtype hctr                 = top_data_buffer[buffer_index+2];
    Dtype hdiff                = top_data_buffer[buffer_index+3];
    Dtype gain_x_all           = top_data_buffer[buffer_index+4];
    Dtype gain_y_all           = top_data_buffer[buffer_index+5];
    Dtype dgx_final_dwctr_all  = top_data_buffer[buffer_index+6];
    Dtype dgy_final_dhctr_all  = top_data_buffer[buffer_index+7];
    Dtype dgx_final_dwdiff_all = top_data_buffer[buffer_index+8];
    Dtype dgy_final_dhdiff_all = top_data_buffer[buffer_index+9];

    //if (gain_x_all > 1e-10 && gain_y_all > 1e-10) {

      bottom_rois += n * 5;
      int roi_batch_ind = (bottom_rois[0] - 1);

      Dtype src_w = bottom_rois[3] - bottom_rois[1] + 1; 
      Dtype src_h = bottom_rois[4] - bottom_rois[2] + 1;
      Dtype src_ctr_x = bottom_rois[1] + 0.5*(src_w-1.0); 
      Dtype src_ctr_y = bottom_rois[2] + 0.5*(src_h-1.0); 

      Dtype dst_ctr_x = bottom_delta_rois[1]; // dx (in fast-rcnn notation) = cx (in here)
      Dtype dst_ctr_y = bottom_delta_rois[2]; // dy (in fast-rcnn notation) = cy (in here) 
      Dtype dst_scl_x = bottom_delta_rois[3]; // dw (in fast-rcnn notation) = sx (in here)
      Dtype dst_scl_y = bottom_delta_rois[4]; // dh (in fast-rcnn notation) = sy (in here) 

      Dtype pred_ctr_x = dst_ctr_x * src_w + src_ctr_x; // dpcx / dcx = src_w
      Dtype pred_ctr_y = dst_ctr_y * src_h + src_ctr_y; // dpcy / dcy = src_h
      Dtype pred_w = exp(dst_scl_x) * src_w;            // dpw  / dsx = src_w * exp(dsx)  
      Dtype pred_h = exp(dst_scl_y) * src_h;            // dph  / dsy = src_h * exp(dsy)  
      
      Dtype roi_start_w = ( (pred_ctr_x - 0.5*(pred_w-1)) - 1 ) * spatial_scale; // drsw / dpcx =       spatial_scale 
                                                                                 // drsw / dpw = -0.5 * spatial_scale
      Dtype roi_start_h = ( (pred_ctr_y - 0.5*(pred_h-1)) - 1 ) * spatial_scale; // drsh / dpcy =       spatial_scale 
                                                                                 // drsh / dph = -0.5 * spatial_scale
      Dtype roi_end_w =   ( (pred_ctr_x + 0.5*(pred_w-1)) - 1 ) * spatial_scale; // drew / dpcx =       spatial_scale 
                                                                                 // drew / dpw =  0.5 * spatial_scale
      Dtype roi_end_h =   ( (pred_ctr_y + 0.5*(pred_h-1)) - 1 ) * spatial_scale; // dreh / dpcy =       spatial_scale 
                                                                                 // dreh / dph =  0.5 * spatial_scale
      assert(roi_end_w - roi_start_w >= 0); 
      assert(roi_end_h - roi_start_h >= 0); 
      
      // drsw / dcx = drsw / dpcx * dpcx / dcx = spatial_scale * src_w
      // drew / dcx = drew / dpcx * dpcx / dcx = spatial_scale * src_w

      // drsh / dcy = drsh / dpcy * dpcy / dcy = spatial_scale * src_h
      // dreh / dcy = dreh / dpcy * dpcy / dcy = spatial_scale * src_h

      // drsw / dsx = drsw / dpw * dpw / dsx = -0.5 * spatial_scale * src_w * exp(dsx) 
      // drew / dsx = drew / dpw * dpw / dsx =  0.5 * spatial_scale * src_w * exp(dsx)
 
      // drsh / dsy = drsh / dph * dph / dsy = -0.5 * spatial_scale * src_h * exp(dsy)
      // dreh / dsy = dreh / dph * dph / dsy =  0.5 * spatial_scale * src_h * exp(dsy) 
 
      // Force malformed ROIs to be 1x1
      Dtype roi_width  = roi_end_w - roi_start_w + 1; // drw / drew =  1 
                                                      // drw / drsw = -1
      Dtype roi_height = roi_end_h - roi_start_h + 1; // drh / dreh =  1 
                                                      // drh / drsh = -1 
      // drw / dcx = drw / drew * drew / dcx + drw / drsw * drsw / dcx = drew / dcx - drsw / dcx
      //           = spatial_scale * src_w - spatial_scale * src_w = 0 
      // drh / dcy = drh / dreh * dreh / dcy + drh / drsh * drsh / dcy = dreh / dcy - drsh / dcy = spatial_scale * src_h - spatial_scale * src_h = 0 
      // drw / dsx = drw / drew * drew / dsx + drw / drsw * drsw / dsx = drew / dsx - drsw / dsx = 0.5 * spatial_scale * src_w * exp(dsx) - (-0.5 * spatial_scale * src_w * exp(dsx)) = spatial_scale * src_w * exp(dsx) 
      // drh / dsy = drh / dreh * dreh / dsy + drh / drsh * drsh / dsy = dreh / dsy - drsh / dsy = 0.5 * spatial_scale * src_h * exp(dsy) - (-0.5 * spatial_scale * src_h * exp(dsy)) = spatial_scale * src_h * exp(dsy) 

      Dtype bin_size_w = roi_width  / static_cast<Dtype>(pooled_width);  // dbw / drw  =  1 / pooled_width
      Dtype bin_size_h = roi_height / static_cast<Dtype>(pooled_height); // dbh / drh  =  1 / pooled_height
      // dbw / dcx = dbw / drw * drw / dcx = 0 
      // dbh / dcy = dbh / drh * drh / dcy = 0
      // dbw / dsx = dbw / drw * drw / dsx = 1 / pooled_width  * spatial_scale * src_w * exp(dsx) 
      // dbh / dsy = dbh / drh * drh / dsy = 1 / pooled_height * spatial_scale * src_h * exp(dsy) 

      Dtype wstart_ = static_cast<Dtype>(pw) * bin_size_w + roi_start_w; // ws = f(rsw, rew)
      Dtype hstart_ = static_cast<Dtype>(ph) * bin_size_h + roi_start_h; // hw = f(rsh, reh)
      Dtype wend_ = static_cast<Dtype>(pw+1) * bin_size_w + roi_start_w; // we = f(rsw, rew)
      Dtype hend_ = static_cast<Dtype>(ph+1) * bin_size_h + roi_start_h; // he = f(rsh, reh)
      // dws / dbw = pw 
      // dhs / dbh = ph 
      // dwe / dbw = (pw+1)
      // dhe / dbh = (ph+1)
                                                                        
      int wstart = static_cast<int>(floor(wstart_));
      int hstart = static_cast<int>(floor(hstart_));
      int wend   = static_cast<int>( ceil(wend_));
      int hend   = static_cast<int>( ceil(hend_));

      // dws / dcx = dws / drsw * drsw / dcx + dws / drew * drew / dcx
      //           = (dws / dbw * dbw / drsw + 1) * drsw / dcx + (dws / dbw * dbw / drew) * drew / dcx
      //           = (pw * 1 / pooled_width * (-1) + 1) * spatial_scale * src_w 
      //           + (pw * 1 / pooled_width * ( 1)    ) * spatial_scale * src_w 
      //           = spatial_scale * src_w
      // dwe / dcx = dwe / drsw * drsw / dcx + dwe / drew * drew / dcx
      //           = (dwe / dbw * dbw / drsw + 1) * drsw / dcx + (dwe / dbw * dbw / drew) * drew / dcx 
      //           = ((pw+1) * 1 / pooled_width * (-1) + 1) * spatial_scale * src_w
      //           + ((pw+1) * 1 / pooled_width * ( 1)    ) * spatial_scale * src_w
      //           = spatial_scale * src_w

      // dws / dsx = dws / drsw * drsw / dsx + dws / drew * drew / dsx 
      //           = (dws / dbw * dbw / drsw + 1) * drsw / dsx + (dws / dbw * dbw / drew) * drew / dsx
      //           = (pw * 1 / pooled_width * (-1) + 1) * (-0.5 * spatial_scale * src_w * exp(dsx))
      //           + (pw * 1 / pooled_width * ( 1)    ) * ( 0.5 * spatial_scale * src_w * exp(dsx))
      //           = (pw * 1 / pooled_width - 0.5) * spatial_scale * src_w * exp(dsx)  
      // dwe / dsx = dwe / drsw * drsw / dsx + dwe / drew * drew / dsx
      //           = (dwe / dbw * dbw / drsw + 1) * drsw / dsx + (dwe / dbw * dbw / drew) * drew / dsx
      //           = ((pw+1) * 1 / pooled_width * (-1) + 1) * (-0.5 * spatial_scale * src_w * exp(dsx)) 
      //           + ((pw+1) * 1 / pooled_width * ( 1)    ) * ( 0.5 * spatial_scale * src_w * exp(dsx)) 
      //           = ((pw+1) * 1 / pooled_width - 0.5) * spatial_scale * src_w * exp(dsx)

      // dws / dcy = spatial_scale * src_h
      // dwe / dcy = spatial_scale * src_h
      // dws / dsy = (   ph  * 1 / pooled_height - 1) * spatial_scale * src_h * exp(dsy)
      // dwe / dsy = ((ph+1) * 1 / pooled_height - 1) * spatial_scale * src_h * exp(dsy)
      /*
      Dtype wctr =  (wend_ + wstart_) * 0.5; // dwctr / dwe = 0.5; dwctr / dws = 0.5
      Dtype hctr =  (hend_ + hstart_) * 0.5; // dhctr / dhe = 0.5; dhctr / dhs = 0.5
      Dtype wdiff = (wend_ - wstart_) + 1;   // dwdiff / dwe = 1; dwdiff / dws = -1
      Dtype hdiff = (hend_ - hstart_) + 1;   // dhdiff / dhe = 1; dhdiff / dhs = -1

      // dwctr  / dcx = dwctr  / dwe * dwe / dcx + dwctr  / dws * dws / dcx = 0.5 * spatial_scale * src_w + 0.5 * spatial_scale * src_w = spatial_scale * src_w 
      // dwdiff / dcx = dwdiff / dwe * dwe / dcx + dwdiff / dws * dws / dcx =   1 * spatial_scale * src_w -  1  * spatial_scale * src_w = 0 
      // dhctr  / dcy = spatial_scale * src_h
      // dhdiff / dcy = 0
  
      // dwctr  / dsx = dwctr / dwe * dwe / dsx + dwctr / dws * dws / dsx 
      //              = 0.5 * ((pw+1)/pooled_width - 0.5) * spatial_scale * src_w * exp(dsx) 
      //              + 0.5 * ( pw   /pooled_width - 0.5) * spatial_scale * src_w * exp(dsx) 
      //              = 0.5 * ((2*pw+1)/pooled_width - 1) * spatial_scale * src_w * exp(dsx)
      //              = ((pw + 0.5) / pooled_width - 0.5) * spatial_scale * src_w * exp(dsx) 
      // dwdiff / dsx = dwdiff / dwe * dwe / dsx + dwdiff / dws * dws / dsx 
      //              = 1  * ((pw+1)/pooled_width - 0.5) * spatial_scale * src_w * exp(dsx)
      //            + (-1) * ( pw   /pooled_width - 0.5) * spatial_scale * src_w * exp(dsx)
      //              = (1 / pooled_width) * spatial_scale * src_w * exp(dsx) 
      // dhctr  / dsy = ((ph + 0.5) / pooled_height - 0.5) * spatial_scale * src_h * exp(dsy)
      // dhdiff / dsy = (1 / pooled_height) * spatial_scale * src_h * exp(dsy) 


      // dgx / dwctr  = (w >= wctr ? 1 : -1)  
      // dgx / dwdiff = 1 
      // dgy / dhctr  = (h >= hctr ? 1 : -1)  
      // dgy / dhdiff = 1
 
      // gx_final = gx / gx_all 
      // dgx_final / dwctr  = ( dgx/dwctr  * gx_all - gx * dgx_all/dwctr  ) / (gx_all)^2 = ( (w >= wctr ? 1 : -1) * gx_all - gx * sum_for_w{ (w >= wctr ? 1 : -1) } ) / gx_all^2
      // dgx_final / dwdiff = ( dgx/dwdiff * gx_all - gx * dgx_all/dwdiff ) / (gx_all)^2 = (       1              * gx_all - gx * sum_for_w{          1           } ) / gx_all^2
      // gy_final = gy / gy_all
      // dgy_final / dhctr  = ...
      // dgy_final / dhdiff = ...

      // dgx_final / dcx = dgx_final / dwctr * dwctr / dcx + dgx_final / dwdiff * dwdiff / dcx
      //                 = ( (w >= wctr ? 1 : -1) * gx_all - gx * sum_for_w{ (w >= wctr ? 1 : -1) } ) / gx_all^2 * spatial_scale * src_w + (...) * 0
      //                 = ( (w >= wctr ? 1 : -1) * gx_all - gx * sum_for_w{ (w >= wctr ? 1 : -1) } ) / gx_all^2 * spatial_scale * src_w 
      // dgy_final / dcy = ( (h >= hctr ? 1 : -1) * gy_all - gy * sum_for_h{ (h >= hctr ? 1 : -1) } ) / gx_all^2 * spatial_scale * src_h
      // dgx_final / dsx = ( (w >= wctr ? 1 : -1) * gx_all - gx * sum_for_w{ (w >= wctr ? 1 : -1) } ) / gx_all^2 * ((pw + 0.5) - 0.5 * pooled_width)  / pooled_width  * spatial_scale * src_w * exp(dsx) + 
      //                   (           1          * gx_all - gx * sum_for_w{         1            } ) / gx_all^2 *     1                              / pooled_width  * spatial_scale * src_w * exp(dsx) 
      // dgy_final / dsy = ( (h >= hctr ? 1 : -1) * gy_all - gy * sum_for_h{ (h >= hctr ? 1 : -1) } ) / gy_all^2 * ((ph + 0.5) - 0.5 * pooled_height) / pooled_height * spatial_scale * src_h * exp(dsy) + 
      //                   (           1          * gy_all - gy * sum_for_h{         1            } ) / gy_all^2 *     1                              / pooled_height * spatial_scale * src_h * exp(dsy) 

      // dg / dcx = dg / dgx_final * dgx_final / dcx + dg / dgy_final * dgy_final / dcx
      //          =   gy_final     * dgx_final / dcx +   gx_final     * 0
      //          =   gy_final     * dgx_final / dcx
      // ... 
      */ 
      // Add roi offsets and clip to input boundaries
      hstart = min(max(hstart, 0), height);
      hend   = min(max(hend, 0), height);
      wstart = min(max(wstart, 0), width);
      wend   = min(max(wend, 0), width);

      // Define an empty pooling region to be zero
      Dtype val_cx = 0, val_cy = 0, val_sx = 0, val_sy = 0; 
      Dtype gain_x = 0, gain_y = 0;  
      Dtype pw_ = static_cast<Dtype>(pw); 
      Dtype ph_ = static_cast<Dtype>(ph);
      Dtype pooled_width_  = static_cast<Dtype>(pooled_width); 
      Dtype pooled_height_ = static_cast<Dtype>(pooled_height);
      bottom_data += (roi_batch_ind * channels + c) * height * width;
      Dtype w_mask = 0, h_mask = 0, coeff_x = 0, coeff_y = 0; 
      for (int h = hstart; h < hend; ++h) {
        for (int w = wstart; w < wend; ++w) {
          int bottom_index = h * width + w;
          Dtype w_ = w, h_ = h;  
          gain_x = wdiff - abs((w_ - wctr));   
          gain_y = hdiff - abs((h_ - hctr));   

          w_mask = w_ >= wctr ? 1 : -1;   
          h_mask = h_ >= hctr ? 1 : -1;  

          //val_cx = val_cx + gain_y / gain_y_all * (w_mask * gain_x_all - gain_x * dgx_final_dwctr_all ) / (gain_x_all*gain_x_all)                            * spatial_scale * src_w * top_diff[index]; 
          //val_cy = val_cy + gain_x / gain_x_all * (h_mask * gain_y_all - gain_y * dgy_final_dhctr_all ) / (gain_y_all*gain_y_all)                            * spatial_scale * src_h * top_diff[index];
          //val_sx = val_sx + gain_y / gain_y_all *((         gain_x_all - gain_x * dgx_final_dwdiff_all) / (gain_x_all*gain_x_all) * (pw_+0.5-0.5*pooled_width) / pooled_width * spatial_scale * src_w * exp(dsx) + 
          //                                        (w_mask * gain_x_all - gain_x * dgx_final_dwctr_all ) / (gain_x_all*gain_x_all) *            1               / pooled_width * spatial_scale * src_w * exp(dsx) ) * top_diff[index]; 
          //val_sy = val_sy + gain_x / gain_x_all *((         gain_y_all - gain_y * dgy_final_dhdiff_all) / (gain_y_all*gain_y_all) * (ph_+0.5-0.5*pooled_height)/ pooled_hidth * spatial_scale * src_h * eyp(dsy) +
          //                                        (h_mask * gain_y_all - gain_y * dgy_final_dhctr_all ) / (gain_y_all*gain_y_all) *            1               / pooled_hidth * spatial_scale * src_h * eyp(dsy) ) * top_diff[index];

          //if (gain_x > 1e-10 && gain_y > 1e-10) {
            coeff_x = bottom_data[bottom_index] * gain_y * spatial_scale * src_w * top_diff[index];
            if (gain_x_all > 1e-10) {coeff_x = coeff_x / (gain_x_all*gain_x_all);} 
            if (gain_y_all > 1e-10) {coeff_x = coeff_x / gain_y_all;}
            val_cx = val_cx +  (w_mask * gain_x_all - gain_x * dgx_final_dwctr_all ) * coeff_x;
            val_sx = val_sx + ((w_mask * gain_x_all - gain_x * dgx_final_dwctr_all ) * (pw_+0.5-0.5*pooled_width_) +
                               (         gain_x_all - gain_x * dgx_final_dwdiff_all)) / pooled_width_ * coeff_x * exp(dst_scl_x);
          
            coeff_y = bottom_data[bottom_index] * gain_x * spatial_scale * src_h * top_diff[index];
            if (gain_y_all > 1e-10) {coeff_y = coeff_y / (gain_y_all*gain_y_all);}
            if (gain_x_all > 1e-10) {coeff_y = coeff_y / gain_x_all;}
            val_cy = val_cy +  (h_mask * gain_y_all - gain_y * dgy_final_dhctr_all ) * coeff_y;
            val_sy = val_sy + ((h_mask * gain_y_all - gain_y * dgy_final_dhctr_all ) * (ph_+0.5-0.5*pooled_height_) + 
                               (         gain_y_all - gain_y * dgy_final_dhdiff_all)) / pooled_height_ * coeff_y * exp(dst_scl_y);
          //}
        }
      }
      /*int*/ buffer_index = n * (channels * pooled_height * pooled_width * 4) + c * (pooled_height * pooled_width * 4) + ph * (pooled_width * 4) + pw * 4; 
      bottom_diff_delta_rois_buffer[buffer_index+0] = val_cx; 
      bottom_diff_delta_rois_buffer[buffer_index+1] = val_cy; 
      bottom_diff_delta_rois_buffer[buffer_index+2] = val_sx;
      bottom_diff_delta_rois_buffer[buffer_index+3] = val_sy;
    //}
  }
}


extern "C"
void inn_ROIWarping_updateGradInputAtomic(THCState *state,
    THCudaTensor *gradInput_data, THCudaTensor *data,   
    THCudaTensor *gradInput_delta_rois, THCudaTensor *delta_rois,
    THCudaTensor *gradInput_delta_rois_buffer,
    THCudaTensor *gradOutput, THCudaTensor *top_data_buffer, 
    THCudaTensor* rois, int W, int H, double spatial_scale)
{
  THAssert(THCudaTensor_nDimension(state, data) == 4);
  THAssert(THCudaTensor_nDimension(state, top_data_buffer) == 5);
  THAssert(THCudaTensor_nDimension(state, rois) == 2 && rois->size[1] == 5);
  THAssert(THCudaTensor_nDimension(state, delta_rois) == 2 && delta_rois->size[1] == 5);
  THAssert(THCudaTensor_nDimension(state, rois) == THCudaTensor_nDimension(state, delta_rois) &&
           rois->size[0] == delta_rois->size[0] &&
           rois->size[1] == delta_rois->size[1]);
  THAssert(THCudaTensor_isContiguous(state, data));
  THAssert(THCudaTensor_isContiguous(state, top_data_buffer));
  THAssert(THCudaTensor_isContiguous(state, rois));
  THAssert(THCudaTensor_isContiguous(state, delta_rois));
  long num_rois = rois->size[0];
  long nInputPlane = data->size[1];
  THCudaTensor_resizeAs(state, gradInput_data, data);
  THCudaTensor_zero(state, gradInput_data);
  THCudaTensor_resizeAs(state, gradInput_delta_rois, delta_rois);
  THCudaTensor_zero(state, gradInput_delta_rois);
  THCudaTensor_resize5d(state, gradInput_delta_rois_buffer, num_rois, nInputPlane, H, W, 4);
  THCudaTensor_zero(state, gradInput_delta_rois_buffer);

  //Backpropagation for data
  long count = THCudaTensor_nElement(state, gradInput_data);
  for (int nth_roi = 0; nth_roi < num_rois; ++nth_roi) {
    ROIWarpBackwardData<float><<<GET_BLOCKS(count), CUDA_NUM_THREADS / 2, 0, THCState_getCurrentStream(state)>>>(
        count,
        THCudaTensor_data(state, top_data_buffer),
        spatial_scale, nInputPlane, data->size[2], data->size[3], H, W, nth_roi,
        THCudaTensor_data(state, rois),
        THCudaTensor_data(state, delta_rois),
        THCudaTensor_data(state, gradOutput), 
        THCudaTensor_data(state, gradInput_data)
        );
  }

  //Backpropagation for delta_roi
  count = THCudaTensor_nElement(state, gradOutput);
  ROIWarpBackwardDeltaROI<float><<<GET_BLOCKS(count), CUDA_NUM_THREADS / 2, 0, THCState_getCurrentStream(state)>>>(
      count,
      THCudaTensor_data(state, top_data_buffer),
      spatial_scale, nInputPlane, data->size[2], data->size[3], H, W, 
      THCudaTensor_data(state, rois),
      THCudaTensor_data(state, delta_rois),
      THCudaTensor_data(state, gradOutput),
      THCudaTensor_data(state, data),
      THCudaTensor_data(state, gradInput_delta_rois_buffer)
      );

  // check for errors
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    printf("error in inn_ROIWarping_updateGradInputAtomic: %s\n", hipGetErrorString(err));
    THError("aborting");
  }
}
